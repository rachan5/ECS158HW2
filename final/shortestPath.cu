#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <R.h>
#include <Rinternals.h>
#include <hip/hip_runtime.h>

//nvcc -c shortestPath.cu -Xcompiler "-fpic" -I/usr/include/R
//R CMD SHLIB shortestPath.o -o shortestPath.so

struct node
{
	int nodeID, ancestor;
	char label[20];
	//nodeType - 0 = root, 1 = internal, 2 = tip
};//node

void setNode(node &phy, int numNodes, int id, int aID, const char * label)
{
	phy.nodeID = id;
	phy.ancestor = aID;
	memset(phy.label, '\0', sizeof(label));
	strcpy(phy.label, label);
}//setNode


__global__ void kernel(node * array, int numNodes, int id1, int id2,
											int * ancestorID1, int * ancestorID2)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < numNodes)
	{
		if (array[idx].nodeID == id1)
		{
			int ancestorIndex = 0;
			node temp = array[idx];
			while (temp.ancestor != 0)
			{
				ancestorID1[ancestorIndex++] = temp.ancestor;
				for (int i=0; i<numNodes; i++)
				{
					if (array[i].nodeID == temp.ancestor)
					{
						temp = array[i];
						break;
					}//if
				}//for
			}//while
		}//if	
		else if (array[idx].nodeID == id2)
		{
			int ancestorIndex = 0;
			node temp = array[idx];
			while (temp.ancestor != 0)
			{
				ancestorID2[ancestorIndex++] = temp.ancestor;
				for (int i=0; i<numNodes; i++)
				{
					if (array[i].nodeID == temp.ancestor)
					{
						temp = array[i];
						break;
					}//if
				}//for
			}//while
		}//if
	
	}//if
}//kernel


int * shortestPath(node * phy, int numNodes, const char * label1, const char * label2)
{
	node * deviceArray;
	int * deviceID1;
	int * deviceID2;
	int * ancestorID1 = new int[numNodes];
	int * ancestorID2 = new int[numNodes];
	float blockSize = 1024; //num threads per block

	//check if invalid query
	node temp1, temp2;
	for (int i=0; i<numNodes; i++)
	{
		ancestorID1[i] = 0;
		ancestorID2[i] = 0;
		if (strcmp(label1, phy[i].label) == 0)
			temp1 = phy[i];
		else if (strcmp(label2, phy[i].label) == 0)
			temp2 = phy[i];
	}//for

	if ((temp1.ancestor == temp2.nodeID) || (temp2.ancestor == temp1.nodeID))
	{
		return 0;
	}//if

	hipMalloc(&deviceArray, sizeof(node) * numNodes);
	hipMalloc(&deviceID1, sizeof(int) * numNodes);
	hipMalloc(&deviceID2, sizeof(int) * numNodes);
	hipMemcpy(deviceArray, phy, sizeof(node) * numNodes, hipMemcpyHostToDevice);	
	hipMemcpy(deviceID1, ancestorID1, sizeof(int) * numNodes, hipMemcpyHostToDevice);
	hipMemcpy(deviceID2, ancestorID2, sizeof(int) * numNodes, hipMemcpyHostToDevice);

	dim3 dimBlock(blockSize);
	dim3 dimGrid(ceil(numNodes/blockSize));

	//map phy to complete tree
	kernel <<< dimGrid, dimBlock >>> (deviceArray, numNodes, temp1.nodeID, temp2.nodeID, deviceID1, deviceID2);
	hipMemcpy(ancestorID1, deviceID1, sizeof(int) * numNodes, hipMemcpyDeviceToHost);
	hipMemcpy(ancestorID2, deviceID2, sizeof(int) * numNodes, hipMemcpyDeviceToHost);
	hipFree(deviceArray);
	hipFree(deviceID1);
	hipFree(deviceID2);

	//find shortest path
	int * path = new int[numNodes];
	int currentPath = ancestorID1[0];
	int pathIndex = 0;
	bool isLCAPath = false;
	for (int i=0; i<numNodes; i++)
	{
		path[i] = 0;
		if (temp1.nodeID == ancestorID2[i])
		{
			for (int j=0; j<i; j++)
				path[j] = ancestorID2[j];
			isLCAPath = true;
			break;
		}//if	
		else if (temp2.nodeID == ancestorID1[i])
		{	
			for (int j=0; j<i; j++)
				path[j] = ancestorID1[j];
			isLCAPath = true;
			break;
		}//else if
	}//for

	if (!isLCAPath)
	{
		for(int i=0; i<numNodes; i++)
		{	
			for (int j=0; j<numNodes; j++)
			{
				if (currentPath == ancestorID2[j])
					break;
				if ((ancestorID2[j] == 0) || (j == numNodes-1))
				{
					path[pathIndex++] = ancestorID1[i];
					currentPath = ancestorID1[i];
					break;
				}//if
			}//for	
		}//for		

		if (pathIndex == 0)	
			path[pathIndex++] = currentPath;
	
		for (int i=0; i<numNodes; i++)
		{
			if (ancestorID2[i] == currentPath)
				break;
			path[pathIndex++] = ancestorID2[i];
		}//for
	}//if

	/*
	for (int i=0; i<numNodes; i++)
	{
		if (path[i] == 0)
			break;
		for (int j=0; j<numNodes; j++)
		{
			if (path[i] == phy[j].nodeID)
			{
				printf("%s ", phy[j].label);
				break;
			}//if
		}//for
	}//for

	printf("\n");
	for (int i=0; i<numNodes; i++)
	{
		if (path[i] == 0)
			break;
		printf("%d ", path[i]);	
	}//for
	printf("\n");
*/
	delete [] ancestorID1;
	delete [] ancestorID2;
	//delete [] path;
	return path;
}//shortestPath


SEXP cudaShortestPath(SEXP nodeIDs, SEXP nodeAncestors, SEXP nodeLabels, SEXP n1, SEXP n2)
{
	nodeIDs = coerceVector(nodeIDs, INTSXP);
	nodeAncestors = coerceVector(nodeAncestors, INTSXP);
	nodeLabels = coerceVector(nodeLabels, STRSXP);
	n1 = coerceVector(n1, STRSXP);
	n2 = coerceVector(n2, STRSXP);
	
	int numNodes = length(nodeIDs);
	node * phy = new node[numNodes];
	
	for (int i=0; i<numNodes; i++)
		setNode(phy[i], numNodes, INTEGER(nodeIDs)[i], INTEGER(nodeAncestors)[i], CHAR(STRING_ELT(nodeLabels, i)));

	//test shortest path
	SEXP Rval;
	PROTECT(Rval = allocVector(INTSXP, numNodes));
	int * path = shortestPath(phy, numNodes, CHAR(STRING_ELT(n1, 0)), CHAR(STRING_ELT(n2, 0))); 
	
	for (int i=0; i<numNodes; i++)
		INTEGER(Rval)[i] = path[i];

	delete [] phy;
	delete [] path;
	UNPROTECT(1);
	return Rval;
}//main


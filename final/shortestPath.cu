#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <R.h>
#include <Rinternals.h>
#include <hip/hip_runtime.h>

//nvcc -c shortestPath.cu -Xcompiler "-fpic" -I/usr/include/R
//R CMD SHLIB shortestPath.o -o shortestPath.so

struct node
{
	int nodeID, ancestor;
	char label[20];
	//nodeType - 0 = root, 1 = internal, 2 = tip
};//node

void setNode(node &phy, int numNodes, int id, int aID, const char * label)
{
	phy.nodeID = id;
	phy.ancestor = aID;
	memset(phy.label, '\0', sizeof(label));
	strcpy(phy.label, label);
}//setNode


__global__ void kernel(node * array, int numNodes, int id, int * ancestorID)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
 
  if (idx < numNodes)
	{
		int ancestorIndex = 0;
  	for(int i=0; i<numNodes; i++)
		{
			if (array[i].nodeID == id)
			{
				node temp = array[i];
				while (temp.ancestor != 0)
				{
					ancestorID[ancestorIndex++] = temp.ancestor;
					for (int j=0; j<numNodes; i++)
					{
						if (array[j].nodeID == temp.ancestor)
						{
							temp = array[j];
							break;
						}//if
					}//for
				}//while
			}//if
		}//for
	}//if
/*	else if (array[idx].nodeID == id2)
		{
			node temp = array[idx];
			while (temp.ancestor != 0)
			{
				for (int i=0; i<numNodes; i++)
				{
					if (ancestorID2[i] != 0)
						continue;
					ancestorID2[i] = temp.ancestor;
				}//for
				for (int i=0; i<numNodes; i++)
				{
					if (array[i].nodeID == temp.ancestor)
					{
						temp = array[i];
						break;
					}//if
				}//for
			}//while
		}//if
	}//if
*/
}//kernel


int * shortestPath(node * phy, int numNodes, const char * label1, const char * label2)
{
	node * deviceArray;
	int * deviceID;
	int * ancestorID1 = new int[numNodes];
	int * ancestorID2 = new int[numNodes];
	float blockSize = 1024; //num threads per block

	//check if invalid query
	node temp1, temp2;
	for (int i=0; i<numNodes; i++)
	{
		ancestorID1[i] = 0;
		ancestorID2[i] = 0;
		if (strcmp(label1, phy[i].label) == 0)
			temp1 = phy[i];
		else if (strcmp(label2, phy[i].label) == 0)
			temp2 = phy[i];
	}//for

	if ((temp1.ancestor == temp2.nodeID) || (temp2.ancestor == temp1.nodeID))
	{
		return 0;
	}//if

	dim3 dimBlock(blockSize);
	dim3 dimGrid(ceil(numNodes/blockSize));

	hipMalloc(&deviceArray, sizeof(node) * numNodes);
	hipMalloc(&deviceID, sizeof(int) * numNodes);
	hipMemcpy(deviceArray, phy, sizeof(node) * numNodes, hipMemcpyHostToDevice);	
	hipMemcpy(deviceID, ancestorID1, sizeof(int) * numNodes, hipMemcpyHostToDevice);

  kernel <<< dimBlock, dimGrid >>> (deviceArray, numNodes, temp1.nodeID, deviceID);
	hipMemcpy(ancestorID1, deviceID, sizeof(int) * numNodes, hipMemcpyDeviceToHost);
  
	int ancestorIndex = 0;
  for(int i=0; i<numNodes; i++)
	{
		if (phy[i].nodeID == temp1.nodeID)
		{
			node temp = phy[i];
			while (temp.ancestor != 0)
			{
				ancestorID1[ancestorIndex++] = temp.ancestor;
				for (int j=0; j<numNodes; i++)
				{
					if (array[j].nodeID == temp.ancestor)
					{
						temp = phy[j];
						break;
					}//if
				}//for
			}//while
		}//if
	}//for
	
	ancestorIndex = 0;
  for(int i=0; i<numNodes; i++)
	{
		if (phy[i].nodeID == temp2.nodeID)
		{
			node temp = phy[i];
			while (temp.ancestor != 0)
			{
				ancestorID2[ancestorIndex++] = temp.ancestor;
				for (int j=0; j<numNodes; i++)
				{
					if (array[j].nodeID == temp.ancestor)
					{
						temp = phy[j];
						break;
					}//if
				}//for
			}//while
		}//if
	}//for
	hipFree(deviceArray);
	hipFree(deviceID);

	//find shortest path
	int * path = new int[numNodes];
	int currentPath = ancestorID1[0];
	int pathIndex = 0;
	bool isLCAPath = false;
	for (int i=0; i<numNodes; i++)
	{
		path[i] = 0;
		if (temp1.nodeID == ancestorID2[i])
		{
			for (int j=0; j<i; j++)
				path[j] = ancestorID2[j];
			isLCAPath = true;
			break;
		}//if	
		else if (temp2.nodeID == ancestorID1[i])
		{	
			for (int j=0; j<i; j++)
				path[j] = ancestorID1[j];
			isLCAPath = true;
			break;
		}//else if
	}//for

	if (!isLCAPath)
	{
		for(int i=0; i<numNodes; i++)
		{	
			for (int j=0; j<numNodes; j++)
			{
				if (currentPath == ancestorID2[j])
					break;
				if ((ancestorID2[j] == 0) || (j == numNodes-1))
				{
					path[pathIndex++] = ancestorID1[i];
					currentPath = ancestorID1[i];
					break;
				}//if
			}//for	
		}//for		

		if (pathIndex == 0)	
			path[pathIndex++] = currentPath;
	
		for (int i=0; i<numNodes; i++)
		{
			if (ancestorID2[i] == currentPath)
				break;
			path[pathIndex++] = ancestorID2[i];
		}//for
	}//if

  for(int i =0; i < numNodes; i++ ){
    printf("%d %d\n", ancestorID1[i], ancestorID2[i]);
    //printf("%d %d %s\n", phy[i].nodeID, phy[i].ancestor, phy[i].label);
  }

/*	
	for (int i=0; i<numNodes; i++)
	{
		if (path[i] == 0)
			break;
		for (int j=0; j<numNodes; j++)
		{
			if (path[i] == phy[j].nodeID)
			{
				printf("%s ", phy[j].label);
				break;
			}//if
		}//for
	}//for

	printf("\n");
	for (int i=0; i<numNodes; i++)
	{
		if (path[i] == 0)
			break;
		printf("%d ", path[i]);	
	}//for
	printf("\n");
*/
	delete [] ancestorID1;
	delete [] ancestorID2;
	//delete [] path;
	return path;
}//shortestPath


extern "C" SEXP cudaShortestPath(SEXP nodeIDs, SEXP nodeAncestors, SEXP nodeLabels, SEXP n1, SEXP n2)
{
	nodeIDs = coerceVector(nodeIDs, INTSXP);
	nodeAncestors = coerceVector(nodeAncestors, INTSXP);
	nodeLabels = coerceVector(nodeLabels, STRSXP);
	n1 = coerceVector(n1, STRSXP);
	n2 = coerceVector(n2, STRSXP);
	
	int numNodes = length(nodeIDs);
	node * phy = new node[numNodes];
	
	for (int i=0; i<numNodes; i++){
		//printf("%d %d %s\n", INTEGER(nodeIDs)[i], INTEGER(nodeAncestors)[i], CHAR(STRING_ELT(nodeLabels,i)));
    setNode(phy[i], numNodes, INTEGER(nodeIDs)[i], INTEGER(nodeAncestors)[i], CHAR(STRING_ELT(nodeLabels, i)));
  }
	//test shortest path
	SEXP Rval;
	PROTECT(Rval = allocVector(INTSXP, numNodes));
	//printf("%s\n ", CHAR(STRING_ELT(n1,0)));
  //printf("%d\n", numNodes);
  int * path = shortestPath(phy, numNodes, CHAR(STRING_ELT(n1, 0)), CHAR(STRING_ELT(n2, 0))); 
	
	for (int i=0; i<numNodes; i++){
		//printf("%d\n", path[i]);
    INTEGER(Rval)[i] = path[i];
  }
	delete [] phy;
	delete [] path;
	UNPROTECT(1);
	return Rval;
}//main

